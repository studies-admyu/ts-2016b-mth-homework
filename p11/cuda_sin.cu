#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/* Number of function values to calculate */
#define RANGE_ELEMENTS_COUNT 20000000
/* Maximum possible CUDA blocks */
#define CUDA_BLOCKS_MAX 65535

/** Calculates function values by using GPU (on device)
  *		@param output - output buffer for calculated values;
  *		@param elements_per_thread - elements to be calculated by one GPU thread.
  */
__global__ void device_sin(double* output, unsigned int elements_per_thread)
{
	/* Pi contant */
	const double const_2pi = 8.0 * atan(1.0);
	/* Cycle counter */
	unsigned int i;

	/* Determine indicies of values to be calculated */
	unsigned int thread_start = (blockIdx.x * blockDim.x + threadIdx.x) * elements_per_thread;
	unsigned int thread_end = thread_start + elements_per_thread;
	
	/* Check whenever the thread is not redundant */
	if (thread_start >= RANGE_ELEMENTS_COUNT) {
		return;
	}
	/* Fix indicies excess if any */
	thread_end = (thread_end <= RANGE_ELEMENTS_COUNT)? thread_end: RANGE_ELEMENTS_COUNT;

	/* Calculate values */
	for (i = thread_start; i < thread_end; ++i) {
		output[i] = sin((const_2pi * i) / (RANGE_ELEMENTS_COUNT - 1));
	}
}

/** Calculates function values by using CPU (on host)
  *		@param output - output buffer for calculated values.
  */
void host_sin(double* output)
{
	/* Pi contant */
	const double const_2pi = 8.0 * atan(1.0);
	/* Cycle counter */
	unsigned int i;

	/* Calculate values */
	for (i = 0; i < RANGE_ELEMENTS_COUNT; ++i) {
		output[i] = sin((const_2pi * i) / (RANGE_ELEMENTS_COUNT - 1));
	}
}

int main(int argc, char* argv[])
{
	/* Whenever to use CUDA flag */
	int use_cuda = 0;
	/* Properly calculated values flag */
	int are_counted_properly = 1;
	/* Cycle counter */
	unsigned int i;

	/* CUDA threads per blocks (user defined) */
	unsigned int cuda_threads_per_block = 1;
	/* CUDA blocks count (to be calculated) */
	unsigned int cuda_blocks_count;
	/* Elements to be calculated per each CUDA thread (to be calculated) */
	unsigned int cuda_elements_per_thread = 0;

	/* Buffer for calculated function values on host (will be calculated by CPU, or copied from GPU memory) */
	double* host_function_values = NULL;
	/* Buffer for calculated function values on host (for checking purposes, will be calculated by CPU) */
	double* host_right_function_values = NULL;
	/* Buffer for calculated function values on device (will be calculated by GPU) */
	double* device_function_values = NULL;
	/* Elapsed calculation time */
	float elapsed_time = 0.0f;

	/* Necessary buffer size for calculated values */
	const unsigned int fv_size = RANGE_ELEMENTS_COUNT * sizeof(double);

	/* CUDA events to count elapsed time */
	hipEvent_t cuda_event_start, cuda_event_stop;
	/* Variable for CUDA errors processing */
	hipError_t cuda_error = hipSuccess;

	/* Check whenever there are proper arguments */
	if ((argc > 1) && (argc < 3)) {
		fprintf(stderr, "ERROR: Bad arguments.\n Usage: cuda_sin [--cuda <threads_per_block>]\n");
		return 1;
	} else if ((argc > 1) && (strncmp(argv[1], "--cuda", 7) != 0)) {
		fprintf(stderr, "ERROR: Bad arguments.\n Usage: cuda_sin [--cuda <threads_per_block>]\n");
		return 1;
	}

	if (argc > 1) {
		/* Switch CUDA usage flag */
		use_cuda = 1;
		/* Try to read the --cuda value */
		if (sscanf(argv[2], "%u", &cuda_threads_per_block) != 1) {
			fprintf(stderr, "ERROR: Bad number of threads_per_block.\n");
			return 1;
		} else if (cuda_threads_per_block == 0) {
			/* Bad threads count value */
			fprintf(stderr, "ERROR: Bad number of threads_per_block.\n");
			return 1;
		}
	}

	/* Calculate CUDA blocks count (must not exceed maximum possible value) and elements to be calculated per each thread */
	do {
		++cuda_elements_per_thread;
		cuda_blocks_count = (RANGE_ELEMENTS_COUNT / (cuda_elements_per_thread * cuda_threads_per_block)) +
			((RANGE_ELEMENTS_COUNT % (cuda_elements_per_thread * cuda_threads_per_block) > 0)? 1: 0);
	} while (cuda_blocks_count > CUDA_BLOCKS_MAX);

	/* Create CUDA events */
	cuda_error = hipEventCreate(&cuda_event_start);
	if (cuda_error != hipSuccess) {
		fprintf(stderr, "ERROR: Unable to create CUDA event.\n");
		return 2;
	}

	cuda_error = hipEventCreate(&cuda_event_stop);
	if (cuda_error != hipSuccess) {
		hipEventDestroy(cuda_event_start);
		fprintf(stderr, "ERROR: Unable to create CUDA event.\n");
		return 2;
	}

	/* Allocate host memory for proper values */
	host_right_function_values = (double*)malloc(fv_size);
	if (!host_right_function_values) {
		hipEventDestroy(cuda_event_start);
		hipEventDestroy(cuda_event_stop);
		fprintf(stderr, "ERROR: Unable to allocate host memory.\n");
		return 3;
	}

	/* Allocate host memory for values to be checked */
	host_function_values = (double*)malloc(fv_size);
	if (!host_function_values) {
		free(host_right_function_values);
		hipEventDestroy(cuda_event_start);
		hipEventDestroy(cuda_event_stop);
		fprintf(stderr, "ERROR: Unable to allocate host memory.\n");
		return 3;
	}

	if (use_cuda) {
		/* Allocate GPU memory for values to be calculated */
		cuda_error = hipMalloc((void**)&device_function_values, fv_size);
		if (cuda_error == hipSuccess) {
			/* Asynchronously note that calculation has started */
			hipEventRecord(cuda_event_start, 0);
			/* Asynchronously run GPU calculations */
			device_sin<<<cuda_blocks_count, cuda_threads_per_block>>>(device_function_values, cuda_elements_per_thread);
			/* Asynchronously note that calculation has been completed */
			hipEventRecord(cuda_event_stop, 0);
			/* Wait for all the asynchronous calls to be executed and proceeded */
			hipEventSynchronize(cuda_event_stop);

			/* Copy GPU calculated values into the host memory*/
			hipMemcpy(host_function_values, device_function_values, fv_size, hipMemcpyDeviceToHost);

			/* Deallocate GPU memory */
			hipFree((void*)device_function_values);
		}
	} else {
		/* Asynchronously note that calculation has started */
		hipEventRecord(cuda_event_start, 0);
		/* Wait for event to be proceeded */
		hipEventSynchronize(cuda_event_start);
		/* Run CPU calculations */
		host_sin(host_function_values);
		/* Asynchronously note that calculation has been completed */
		hipEventRecord(cuda_event_stop, 0);
		/* Wait for event to be proceeded */
		hipEventSynchronize(cuda_event_stop);
	}

	/* Caclulate proper values on host */
	host_sin(host_right_function_values);
	/* Check caclulated values to be proper ones */
	for (i = 0; i < RANGE_ELEMENTS_COUNT; ++i) {
		if (abs(host_function_values[i] - host_right_function_values[i]) > 1e-10) {
			are_counted_properly = 0;
			break;
		}
	}

	if (cuda_error == hipSuccess) {
		/* Everything was ok with CUDA - calculate elapsed time in ms */
		hipEventElapsedTime(&elapsed_time, cuda_event_start, cuda_event_stop);
		/* Print information about performed calculations */
		fprintf(
			stdout, "Calculated %s %u values on %s. Time spent: %.02f ms.\n",
			(are_counted_properly)? "properly": "UNproperly",
			RANGE_ELEMENTS_COUNT, (use_cuda)? "device": "host", elapsed_time
		);
	} else {
		/* Something went wrong - just notify the user, cleanup follows */
		fprintf(stderr, "ERROR: Unable to allocate device memory.\n");
	}

	/* Cleanup */
	free(host_function_values);
	free(host_right_function_values);
	
	hipEventDestroy(cuda_event_start);
	hipEventDestroy(cuda_event_stop);

	/* If CUDA commands executed successfully - then everything is ok */
	return (cuda_error == hipSuccess)? 0: 3;
}
